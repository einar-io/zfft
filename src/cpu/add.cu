
#include <hip/hip_complex.h>
#include <vector>
#include "../utils.cuh"

using namespace std;

namespace cpu
{

    // This does not need to be
    void add_pinned(int nElements, hipComplex *h_s, hipComplex *h_p, hipComplex *h_q, hipComplex *dh_s = nullptr, hipComplex *d_p = nullptr, hipComplex *d_q = nullptr, hipComplex *d_s = nullptr)
    {
        for (int i = 0; i < nElements; i++)
        {
            h_s[i] = hipCaddf(h_p[i], h_q[i]);
        }
        return;
    }

    // This is simle
    vector<hipComplex> add_simple(vector<hipComplex> &p, vector<hipComplex> &q)
    {
        ASSERT_MSG(p.size() == q.size(), "The two vectors must have same length.");
        vector<hipComplex> s;
        s.resize(p.size());

        for (int i = 0; i < p.size(); i++)
        {
            s[i] = hipCaddf(p[i], q[i]);
        }

        return s;
    }

}