#include <hip/hip_runtime_api.h>
#include <vector>
#include "../utils.cuh"
#include <hip/hip_complex.h>
#include "gpu.cuh"
#include "hip/driver_types.h"
#include "../include/hip/hip_runtime_api.h"

namespace gpu
{
    using namespace std;
    /*
    void mul_pinned()
    {

        // Phase 1: Allocation

        // Create non-default stream for Async
        hipStream_t stream1;
        checkCudaErrors(hipStreamCreate(&stream1));

        hipComplex *h_p, *h_q, *h_s, *d_p, *d_q, *d_s, *dh_s;

        checkCudaErrors(hipHostMalloc((void **)&h_p, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&h_q, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&h_s, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&dh_s, nBytes));

        checkCudaErrors(hipMalloc((void **)&d_p, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_q, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_s, nBytes));

        // memset
        checkCudaErrors(hipMemset((void *)d_p, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_q, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_s, 0, nBytes));

        // Values
        for (int i = 0; i < nElements; i++)
        {
            h_p[i] = make_hipComplex(float(i), float(i));
            h_q[i] = hipConjf(h_p[i]);
            h_s[i] = hipCaddf(h_p[i], h_q[i]);
            // printf("s[%i]: %f + %fi\n", i, h_s[i].x, h_s[i].y);
        }
        // (1x^2 + 1x + 1) * (1x^2 + 1x + 1) = 1x^4 + 2x^3 + 3x^2 + 2x + 1
        // [1,1,1] * [1,1,1] = [1,2,3,2,1]
        h_p[0] = make_hipComplex(float(1), float(0));
        h_p[1] = make_hipComplex(float(1), float(0));
        h_p[2] = make_hipComplex(float(1), float(0));
        h_p[3] = make_hipComplex(float(0), float(0));
        h_p[4] = make_hipComplex(float(0), float(0));
        h_p[5] = make_hipComplex(float(0), float(0));
        h_p[6] = make_hipComplex(float(0), float(0));
        h_p[7] = make_hipComplex(float(0), float(0));

        h_q[0] = make_hipComplex(float(1), float(0));
        h_q[1] = make_hipComplex(float(1), float(0));
        h_q[2] = make_hipComplex(float(1), float(0));
        h_q[3] = make_hipComplex(float(0), float(0));
        h_q[4] = make_hipComplex(float(0), float(0));
        h_q[5] = make_hipComplex(float(0), float(0));
        h_q[6] = make_hipComplex(float(0), float(0));
        h_q[7] = make_hipComplex(float(0), float(0));

        // Phase 2: Memcpy: Host-to-Device
        checkCudaErrors(hipMemcpyAsync(d_p, h_p, nBytes, hipMemcpyHostToDevice, stream1));
        checkCudaErrors(hipMemcpyAsync(d_q, h_q, nBytes, hipMemcpyHostToDevice, stream1));

        // Phase 3: Execute
        // set kernel launch configuration

        // start
        dim3 blockdim = dim3(32, 1);
        dim3 griddim = dim3(CEIL_DIV(nElements, blockdim.x), 1);
        add_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, (hipComplex *)d_s, (hipComplex *)d_p, (hipComplex *)d_q);
        checkCudaErrors(hipPeekAtLastError());

        mul_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, log_n, (hipComplex *)d_s, (hipComplex *)d_p, (hipComplex *)d_q);
        checkCudaErrors(hipPeekAtLastError());
        // end

    dim3 blockdim = dim3(32, 1);
    dim3 griddim = dim3(CEIL_DIV(32, blockdim.x), 1);
    // fftGPU<<<griddim, blockdim, 0, stream1>>>(4, 2, (hipComplex *)d_p);
    // mul_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, log_n, (hipComplex *)d_s, (hipComplex *)d_p, (hipComplex *)d_q);
    // pad
    gpu::fft<<<griddim, blockdim, 0, stream1>>>(8, 3, (hipComplex *)d_p);
    checkCudaErrors(hipPeekAtLastError());
    gpu::fft<<<griddim, blockdim, 0, stream1>>>(8, 3, (hipComplex *)d_q);
    checkCudaErrors(hipPeekAtLastError());
    // sync
    gpu::pvMul<<<griddim, blockdim, 0, stream1>>>(8, d_s, d_p, d_q);
    checkCudaErrors(hipPeekAtLastError());
    // ifft
    gpu::fft<<<griddim, blockdim, 0, stream1>>>(8, 3, (hipComplex *)d_s, true);
    checkCudaErrors(hipPeekAtLastError());

    // Phase 4: Memcpy: Device-to-Host
    checkCudaErrors(hipMemcpyAsync(dh_s, d_s, nBytes, hipMemcpyDeviceToHost, stream1));

    // Phase 6:  Verification
    printf("Verify\n");
    checkCudaErrors(hipStreamSynchronize(stream1));
    for (int i = 0; i < 8; i++)
    {
        printf("s[%i]: %f + %fi\n", i, dh_s[i].x, dh_s[i].y);
    }
    bool verified = verify(nElements, dh_s, h_s);
    printf("Verification %s.\n", verified ? "succeeded" : "failed");

    // Phase 7: Deallocation
    checkCudaErrors(hipHostFree(h_p));
    checkCudaErrors(hipHostFree(h_q));
    checkCudaErrors(hipHostFree(h_s));
    checkCudaErrors(hipHostFree(dh_s));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_q));
    checkCudaErrors(hipFree(d_s));
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipDeviceReset());
}
*/

    vector<hipComplex> mul_simple(const vector<hipComplex> &p_orig, const vector<hipComplex> &q_orig)
    {
        vector<hipComplex> p(p_orig.begin(), p_orig.end());
        vector<hipComplex> q(q_orig.begin(), q_orig.end());

        ASSERT_MSG(is_power_of_two(p.size()), "Must be a power of two.");
        ASSERT_MSG(p.size() == q.size(), "Must have same size.");

        int old = p.size();
        // pad
        int n = p.size() + q.size();
        int log_n = log_2(n);
        ASSERT_MSG((1 << log_n) == n, "2^log_n = n.");

        p.resize(n);
        q.resize(n);

        hipComplex zero = make_hipComplex(0.0f, 0.0f);
        for (int i = old; i < p.size(); i++)
        {
            p[i] = zero;
            q[i] = zero;
        }

        vector<hipComplex> s;
        s.resize(n);

        // Phase 0: Sizing
        int nElements = n;
        int nBytes = n * sizeof(hipComplex);

        // Phase 1: Allocation

        // Create non-default stream for Async
        hipStream_t stream1;
        checkCudaErrors(hipStreamCreate(&stream1));

        hipComplex *h_p, *h_q, *h_s, *d_p, *d_q, *d_s, *dh_s;

        h_p = p.data();
        h_q = q.data();
        dh_s = s.data();

        checkCudaErrors(hipMalloc((void **)&d_p, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_q, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_s, nBytes));

        // memset
        checkCudaErrors(hipMemset((void *)d_p, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_q, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_s, 0, nBytes));

        // Phase 2: Memcpy: Host-to-Device
        checkCudaErrors(hipMemcpyAsync(d_p, h_p, nBytes, hipMemcpyHostToDevice, stream1));
        checkCudaErrors(hipMemcpyAsync(d_q, h_q, nBytes, hipMemcpyHostToDevice, stream1));

        // Phase 3: Execute
        // set kernel launch configuration

        dim3 blockdim = dim3(32, 1);
        dim3 griddim = dim3(CEIL_DIV(32, blockdim.x), 1);

        gpu::fft<<<griddim, blockdim, 0, stream1>>>(n, log_n, (hipComplex *)d_p);
        checkCudaErrors(hipPeekAtLastError());
        gpu::fft<<<griddim, blockdim, 0, stream1>>>(n, log_n, (hipComplex *)d_q);
        checkCudaErrors(hipPeekAtLastError());
        // sync
        gpu::pvMul<<<griddim, blockdim, 0, stream1>>>(n, d_s, d_p, d_q);
        checkCudaErrors(hipPeekAtLastError());
        // ifft
        gpu::fft<<<griddim, blockdim, 0, stream1>>>(n, log_n, (hipComplex *)d_s, true);
        checkCudaErrors(hipPeekAtLastError());

        // Phase 4: Memcpy: Device-to-Host
        checkCudaErrors(hipMemcpyAsync(dh_s, d_s, nBytes, hipMemcpyDeviceToHost, stream1));

        checkCudaErrors(hipStreamSynchronize(stream1));

        // Phase 7: Deallocation
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_q));
        checkCudaErrors(hipFree(d_s));
        checkCudaErrors(hipStreamDestroy(stream1));
        checkCudaErrors(hipDeviceReset());

        return s;
    }
}