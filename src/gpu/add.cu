#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../utils.cuh"
#include "../verify.cu"
#include "gpu.cuh"
#include <vector>
#include <hip/hip_complex.h>

#include "hip/driver_types.h"
#include "../include/hip/hip_runtime_api.h"

namespace gpu
{
    using namespace std;
    // For performance
    void add_pinned(int n, hipComplex *h_s, hipComplex *h_p, hipComplex *h_q, hipComplex *dh_s, hipComplex *d_p, hipComplex *d_q, hipComplex *d_s)
    {

        // Phase 0: Sizing
        // const unsigned int log_n = log2;
        const unsigned int nElements = n;
        const size_t nBytes = nElements * sizeof(hipComplex);

        // Phase 1: Allocation

        // Create non-default stream for Async
        hipStream_t stream1;
        checkCudaErrors(hipStreamCreate(&stream1));

        /*
                hipComplex *h_p, *h_q, *h_s, *d_p, *d_q, *d_s, *dh_s;

                checkCudaErrors(hipHostMalloc((void **)&h_p, nBytes));
                checkCudaErrors(hipHostMalloc((void **)&h_q, nBytes));
                checkCudaErrors(hipHostMalloc((void **)&h_s, nBytes));
                checkCudaErrors(hipHostMalloc((void **)&dh_s, nBytes));

                checkCudaErrors(hipMalloc((void **)&d_p, nBytes));
                checkCudaErrors(hipMalloc((void **)&d_q, nBytes));
                checkCudaErrors(hipMalloc((void **)&d_s, nBytes));

                // memset
                checkCudaErrors(hipMemset((void *)d_p, 0, nBytes));
                checkCudaErrors(hipMemset((void *)d_q, 0, nBytes));
                checkCudaErrors(hipMemset((void *)d_s, 0, nBytes));
                */

        // Phase 2: Memcpy: Host-to-Device
        checkCudaErrors(hipMemcpyAsync(d_p, h_p, nBytes, hipMemcpyHostToDevice, stream1));
        checkCudaErrors(hipMemcpyAsync(d_q, h_q, nBytes, hipMemcpyHostToDevice, stream1));

        // Phase 3: Execute
        // set kernel launch configuration
        dim3 blockdim = dim3(256, 1);
        dim3 griddim = dim3(CEIL_DIV(nElements, blockdim.x), 1);
        add_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, (hipComplex *)d_s, (hipComplex *)d_p, (hipComplex *)d_q);
        checkCudaErrors(hipPeekAtLastError());

        // Phase 4: Memcpy: Device-to-Host
        checkCudaErrors(hipMemcpyAsync(dh_s, d_s, nBytes, hipMemcpyDeviceToHost, stream1));

        checkCudaErrors(hipStreamSynchronize(stream1));
        checkCudaErrors(hipStreamDestroy(stream1));
        return;

        /*
                // Phase 6:  Verification
                printf("Verify\n");
                checkCudaErrors(hipStreamSynchronize(stream1));
                for (int i = 0; i < 8; i++)
                {
                    printf("s[%i]: %f + %fi\n", i, dh_s[i].x, dh_s[i].y);
                }
                bool verified = verify(nElements, dh_s, h_s);
                printf("Verification %s.\n", verified ? "succeeded" : "failed");

                // Phase 7: Deallocation
                checkCudaErrors(hipHostFree(h_p));
                checkCudaErrors(hipHostFree(h_q));
                checkCudaErrors(hipHostFree(h_s));
                checkCudaErrors(hipHostFree(dh_s));
                checkCudaErrors(hipFree(d_p));
                checkCudaErrors(hipFree(d_q));
                checkCudaErrors(hipFree(d_s));
                checkCudaErrors(hipStreamDestroy(stream1));
                checkCudaErrors(hipDeviceReset());

                return verified;
        */
    }

    // For easing testing
    vector<hipComplex> add_simple(vector<hipComplex> &p, vector<hipComplex> &q)
    //(int n, hipComplex *h_s, hipComplex *h_p, hipComplex *h_q, hipComplex *dh_s, hipComplex *d_p, hipComplex *d_q, hipComplex *d_s)
    {

        // Phase 0: Sizing
        // const unsigned int log_n = log2;
        ASSERT_MSG(p.size() == q.size(), "The two vectors must have same length.");
        const unsigned int nElements = p.size();
        const size_t nBytes = nElements * sizeof(hipComplex);

        vector<hipComplex> s;
        s.resize(p.size());

        // Phase 1: Allocation

        // Create non-default stream for Async
        hipStream_t stream1;
        checkCudaErrors(hipStreamCreate(&stream1));

        hipComplex *h_p, *h_q, *h_s, *d_p, *d_q, *d_s, *dh_s;

        /*
        checkCudaErrors(hipHostMalloc((void **)&h_p, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&h_q, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&h_s, nBytes));
        checkCudaErrors(hipHostMalloc((void **)&dh_s, nBytes));
        */

        h_p = p.data();
        h_q = q.data();
        dh_s = s.data();

        checkCudaErrors(hipMalloc((void **)&d_p, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_q, nBytes));
        checkCudaErrors(hipMalloc((void **)&d_s, nBytes));

        // memset
        checkCudaErrors(hipMemset((void *)d_p, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_q, 0, nBytes));
        checkCudaErrors(hipMemset((void *)d_s, 0, nBytes));

        // Phase 2: Memcpy: Host-to-Device
        checkCudaErrors(hipMemcpyAsync(d_p, h_p, nBytes, hipMemcpyHostToDevice, stream1));
        checkCudaErrors(hipMemcpyAsync(d_q, h_q, nBytes, hipMemcpyHostToDevice, stream1));

        // Phase 3: Execute
        // set kernel launch configuration
        dim3 blockdim = dim3(256, 1);
        dim3 griddim = dim3(CEIL_DIV(nElements, blockdim.x), 1);
        add_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, (hipComplex *)d_s, (hipComplex *)d_p, (hipComplex *)d_q);
        checkCudaErrors(hipPeekAtLastError());

        // Phase 4: Memcpy: Device-to-Host
        checkCudaErrors(hipMemcpyAsync(dh_s, d_s, nBytes, hipMemcpyDeviceToHost, stream1));

        checkCudaErrors(hipStreamSynchronize(stream1));

        // Phase 7: Deallocation
        // checkCudaErrors(hipHostFree(h_p));
        // checkCudaErrors(hipHostFree(h_q));
        // checkCudaErrors(hipHostFree(h_s));
        // checkCudaErrors(hipHostFree(dh_s));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_q));
        checkCudaErrors(hipFree(d_s));
        checkCudaErrors(hipStreamDestroy(stream1));
        checkCudaErrors(hipDeviceReset());

        return s;
    }

}