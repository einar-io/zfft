#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <cstdlib>
#include <cassert>
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions

#define N 1024
#define NDEBUG true
#define assertm(exp, msg) assert(((void)msg, exp))

// hipComplex is hipFloatComplex
typedef hipComplex CoefT;
typedef CoefT Poly[N];

__host__ __device__ static __inline__ bool cuCeqf(hipFloatComplex x,
                                                  hipFloatComplex y)
{
    return hipCrealf(x) == hipCrealf(y) && hipCimagf(x) == hipCimagf(y);
}

__global__ void add_kernel(int n, CoefT *dst, const CoefT *src)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dst[i] = hipCaddf(dst[i], src[i]);
}

int main()
{

    Poly p, q, s, d_p, d_q, d_s;

    for (int i = 0; i < N; i++)
    {
        p[i] = make_hipComplex(float(i), float(i));
        q[i] = hipConjf(p[i]);
        s[i] = hipCaddf(p[i], q[i]);
    }

    int n = 1 << 10;
    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);

    // async
    add_kernel<<<blocks, threads>>>(N, d_s, d_p);
    checkCudaErrors(hipPeekAtLastError());

    for (int i = 0; i < N; i++)
    {
        assertm(cuCeqf(s[i], d_s[i]), "CPU and GPU results differ.");
    }

    std::exit(EXIT_SUCCESS);
}
