#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <cstdlib>
#include <cassert>
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions

#define CEIL_DIV(x, y) (((x) + ((y)-1)) / (y))

#define N 1024
#define DEBUG true
#define assertm(exp, msg) assert(((void)msg, exp))

// hipComplex is hipFloatComplex
typedef hipComplex CoefT;
// typedef CoefT Poly[N];

__host__ __device__ static __inline__ bool cuCeqf(hipFloatComplex x,
                                                  hipFloatComplex y)
{
    // todo: better float comparison
    return hipCrealf(x) == hipCrealf(y) && hipCimagf(x) == hipCimagf(y);
}

__host__ bool verify(const unsigned int n, const CoefT *h_s, const CoefT *d_s)
{
    for (int i = 0; i < n; i++)
    {
        bool compare = cuCeqf(h_s[i], d_s[i]);
        assertm(compare, "CPU and GPU results differ.");
        if (!compare)
            return false;
    }
    return true;
}

__global__ void add_kernel(int n, CoefT *dst, const CoefT *src)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dst[i] = hipCaddf(dst[i], src[i]);
}

int main()
{
    // output device info and transfer size
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, 0));
    printf("\nDevice: %s (device memory: %.1f GiB)\n", prop.name, prop.totalGlobalMem / double(1 << 30));

    // Phase 0: Sizing
    const unsigned int nElements = N;
    const size_t nBytes = nElements * sizeof(hipComplex);
    // printf("poly: %i", sizeof(Poly));
    assertm(sizeof(CoefT) == 2 * sizeof(float), "size of complex");

    printf("Transfer size (MB): %d\n", nBytes / (1024 * 1024));

    // Phase 1: Allocation

    // Create non-default stream for Async
    hipStream_t stream1;
    checkCudaErrors(hipStreamCreate(&stream1));

    CoefT *h_p, *h_q, *h_s, *d_p, *d_q, *d_s, *dh_s;

    checkCudaErrors(hipHostMalloc((void **)&h_p, nBytes));
    checkCudaErrors(hipHostMalloc((void **)&h_q, nBytes));
    checkCudaErrors(hipHostMalloc((void **)&h_s, nBytes));
    checkCudaErrors(hipHostMalloc((void **)&dh_s, nBytes));

    checkCudaErrors(hipMalloc((void **)&d_p, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_q, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_s, nBytes));

    // memset
    checkCudaErrors(hipMemset((void *)d_p, 0, nBytes));
    checkCudaErrors(hipMemset((void *)d_q, 0, nBytes));
    checkCudaErrors(hipMemset((void *)d_s, 0, nBytes));

    // Values
    for (int i = 0; i < nElements; i++)
    {
        h_p[i] = make_hipComplex(float(i), float(i));
        h_q[i] = hipConjf(h_p[i]);
        h_s[i] = hipCaddf(h_p[i], h_q[i]);
        // printf("s[%i]: %f + %fi\n", i, h_s[i].x, h_s[i].y);
    }

    // Phase 2: Memcpy: Host-to-Device
    checkCudaErrors(hipMemcpyAsync(d_p, h_p, nBytes, hipMemcpyHostToDevice, stream1));
    checkCudaErrors(hipMemcpyAsync(d_q, h_q, nBytes, hipMemcpyHostToDevice, stream1));

    // Phase 3: Execute
    // set kernel launch configuration
    dim3 blockdim = dim3(32, 1);
    dim3 griddim = dim3(CEIL_DIV(nElements, blockdim.x), 1);
    add_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, (CoefT *)d_s, (CoefT *)d_p);
    checkCudaErrors(hipPeekAtLastError());
    add_kernel<<<griddim, blockdim, 0, stream1>>>(nElements, (CoefT *)d_s, (CoefT *)d_q);
    checkCudaErrors(hipPeekAtLastError());

    // Phase 4: Memcpy: Device-to-Host
    checkCudaErrors(hipMemcpyAsync(dh_s, d_s, nBytes, hipMemcpyDeviceToHost, stream1));

    // Phase 6:  Verification
    checkCudaErrors(hipStreamSynchronize(stream1));
    printf("Verify\n");
    bool verified = verify(nElements, dh_s, h_s);
    printf("Verification %s.\n", verified ? "succeeded" : "failed");

    // Phase 7: Deallocation
    checkCudaErrors(hipHostFree(h_p));
    checkCudaErrors(hipHostFree(h_q));
    checkCudaErrors(hipHostFree(h_s));
    checkCudaErrors(hipHostFree(dh_s));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_q));
    checkCudaErrors(hipFree(d_s));
    checkCudaErrors(hipStreamDestroy(stream1));

    std::exit(verified ? EXIT_SUCCESS : EXIT_FAILURE);
}
